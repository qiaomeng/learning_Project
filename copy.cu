#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>

#define THREAD_NUM 512

#define MATRIX_SIZE 1000

const int blocks_num = (MATRIX_SIZE + THREAD_NUM - 1) / THREAD_NUM;

void printDeviceProp(const hipDeviceProp_t &prop)
{
	printf("Device Name : %s.\n", prop.name);
	printf("totalFlbalMem : %d.\n", prop.totalGlobalMem);
	printf("sharedMemPerBlock : %d.\n", prop.regsPerBlock);
	printf("regsPerBlock : %d.\n", prop.regsPerBlock);
	printf("warpSize : %d.\n", prop.warpSize);
	printf("memPitch : %d.\n", prop.memPitch);
	printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
	printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("totalConstMem : %d.\n", prop.totalConstMem);
	printf("major.minor : %d.%d.\n", prop.major, prop.minor);
	printf("clockRate : %d.\n", prop.clockRate);
	printf("textureAlignment : %d.\n", prop.textureAlignment);
	printf("deviceOverlap : %d.\n", prop.deviceOverlap);
	printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

bool InitCUDA()
{
	int count;
	hipGetDeviceCount(&count);

	if(count == 0){
		fprintf(stderr, "There is no device.\n");

		return false;
	}

	int i;

	for(i = 0; i < count; i++)
	{
		hipDeviceProp_t prop;
		int status;
		status = hipGetDeviceProperties(&prop, i);
		//printDeviceProp(prop);

		if(status == hipSuccess){
			if(prop.major >= 1){
				break;
			}
		}
	}
	printf("Total %d GPU\n", count);
	if(i == count){
		fprintf(stderr, "There is no device supporting cuda 1.x.\n");
		return false;
	}

	hipSetDevice(i+1);

	return true;
}

void matgen(float* a, int n)
{
	int i, j;
	for(i = 0; i < n; i++){
		for(j = 0; j < n; j++){
			a[i * n + j] = (float)rand() / RAND_MAX + (float)rand() / (RAND_MAX * RAND_MAX);

		}
	}
}

__global__ static void matMultCUDA(const float* a, const float* b, float* c, int n, clock_t* time, int *e)
{
	extern __shared__ float data[];
	const int tid = threadIdx.x;
	const int row = blockIdx.x;
	int i, j;
	//for(i = tid; i < n; i += THREAD_NUM){
	for(i = tid; i < n; i += blockDim.x){
		data[i] = a[row * n + i];
	}
	__syncthreads();
	clock_t start;

	if(tid == 0)
		*e = blockDim.x;
		time[row] = clock();



	//for(j = tid; j < n; j += THREAD_NUM){
	for(j = tid; j < n; j += blockDim.x){
		float t = 0;
		float y = 0;
		for(i = 0; i < n; i++){
			float r ;
			y -= data[i] * b[i * n + j];
			r = t - y;
			y = (r - t) + y;
			t = r;
		}
		c[row * n + j] = t;
	}
	if(tid == 0){
		time[row + blocks_num] = clock();
	}
}

int main()
{
	if(!InitCUDA())
		return 0;

	float *a, *b, *c, *d;
	int *e;

	int n = MATRIX_SIZE;

	a = (float*)malloc(sizeof(float) * n * n);
	b = (float*)malloc(sizeof(float) *n * n);
	c = (float*)malloc(sizeof(float) * n * n);
	d = (float*)malloc(sizeof(float) * n * n);
	e = (int*)malloc(sizeof(int) * n * n);
	srand(0);
	matgen(a, n);
	matgen(b, n);

	float *cuda_a, *cuda_b, *cuda_c;
	int *cuda_e;
	clock_t* time;

	size_t pitch_a, pitch_b, pitch_c;
	hipMallocPitch((void**) &cuda_a, &pitch_a, sizeof(float) * n, n);
	hipMallocPitch((void**)&cuda_b, &pitch_b, sizeof(float) * n, n);
	hipMallocPitch((void**)&cuda_c, &pitch_c, sizeof(float) * n, n);
	/*
	hipMalloc((void**)&cuda_a, sizeof(float) * n * n);
	hipMalloc((void**)&cuda_b, sizeof(float) * n * n);
	hipMalloc((void**)&cuda_c, sizeof(float) * n * n);
	*/
	hipMalloc((void**)&cuda_e, sizeof(int));
	hipMalloc((void**)&time, sizeof(clock_t) * n * 2);

	hipMemcpy(cuda_a, a, sizeof(float) * n * n, hipMemcpyHostToDevice);
	hipMemcpy(cuda_b, b, sizeof(float) * n * n, hipMemcpyHostToDevice);
	//matMultCUDA<<<blocks_num, THREAD_NUM, 0>>>(cuda_a, cuda_b, cuda_c, n, time);
	matMultCUDA<<<n, THREAD_NUM, sizeof(float) * n >>> (cuda_a, cuda_b, cuda_c, n, time, cuda_e);

	clock_t time_use[blocks_num * 2];

	hipMemcpy(c, cuda_c, sizeof(float) * n * n, hipMemcpyDeviceToHost);
	hipMemcpy(e, cuda_e, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&time_use, time, sizeof(clock_t) * 2 * blocks_num, hipMemcpyDeviceToHost);

	hipFree(cuda_a);
	hipFree(cuda_b);
	hipFree(cuda_c);
	hipFree(time);
	printf("sizeof(c)= %d\n",sizeof(c));
	printf("blockDim.x = %d\n", *e);

	for(int i = 0; i < 10; i++)
		printf("%f ", c[i]);
	
	clock_t min_start, max_end;
	min_start = time_use[0];
	max_end = time_use[blocks_num];
	for(int i = 1; i < blocks_num; i++){
		if(min_start > time_use[i])
			min_start = time_use[i];
		if(max_end < time_use[i + blocks_num])
			max_end = time_use[i + blocks_num];
	}
	clock_t  final_time = max_end - min_start;
	for(int i = 0; i < n; i++){
		for(int j = 0; j < n; j++){
			double t = 0;
			for(int k = 0; k < n; k++){
				t += a[i * n + k] * b[k * n + j];
			}
			d[i * n + j] = t;
		}
	}

	float max_err = 0;
	float average_err = 0;
	for(int i = 0; i < n; i++){
		for(int j = 0; j < n; j++){
			if(d[i * n + j] != 0){
				float err = fabs((c[i * n + j] - d[i * n + j]) / d[i * n + j]);
				if(max_err < err)
					max_err = err;
				average_err += err;
			}
		}
	}
	printf("Max error: %g Average error: %g\n", max_err, average_err / (n*n));

	printf("gputime: %d\n", final_time);
	return 0;
}
