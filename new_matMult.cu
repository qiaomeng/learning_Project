#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>

#define NUMTHREADS 512 

#define MATRIX_SIZE 1000

void printDeviceProp(const hipDeviceProp_t &prop)
{
	printf("Device Name : %s.\n", prop.name);
	printf("totalFlbalMem : %d.\n", prop.totalGlobalMem);
	printf("sharedMemPerBlock : %d.\n", prop.regsPerBlock);
	printf("regsPerBlock : %d.\n", prop.regsPerBlock);
	printf("warpSize : %d.\n", prop.warpSize);
	printf("memPitch : %d.\n", prop.memPitch);
	printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
	printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("totalConstMem : %d.\n", prop.totalConstMem);
	printf("major.minor : %d.%d.\n", prop.major, prop.minor);
	printf("clockRate : %d.\n", prop.clockRate);
	printf("textureAlignment : %d.\n", prop.textureAlignment);
	printf("deviceOverlap : %d.\n", prop.deviceOverlap);
	printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

bool InitCUDA()
{
	int count;
	hipGetDeviceCount(&count);

	if(count == 0){
		fprintf(stderr, "There is no device.\n");

		return false;
	}

	int i;

	for(i = 0; i < count; i++)
	{
		hipDeviceProp_t prop;
		int status;
		status = hipGetDeviceProperties(&prop, i);
		//printDeviceProp(prop);

		if(status == hipSuccess){
			if(prop.major >= 1){
				break;
			}
		}
	}
	printf("Total %d GPU\n", count);
	if(i == count){
		fprintf(stderr, "There is no device supporting cuda 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	return true;
}

void matgen(float* a, int n)
{
	int i, j;
	for(i = 0; i < n; i++){
		for(j = 0; j < n; j++){
			a[i * n + j] = (float)rand() / RAND_MAX + (float)rand() / (RAND_MAX * RAND_MAX);

		}
	}
}

__global__ static void matMul

int main()
{
	if(!InitCUDA())
		return 0;

	float *a, *b, *c, *d, *e;

	int n = MATRIX_SIZE;

	a = (float*)malloc(sizeof(float) * n * n);
	b = (float*)malloc(sizeof(float) *n * n);
	c = (float*)malloc(sizeof(float) * n * n);
	d = (float*)malloc(sizeof(float) * n * n);
	srand(0);
	matgen(a, n);
	matgen(b, n);

	for(int i = 0; i < 10; i++)
		printf("%f ", a[i]);
	printf("\n");

	float *cuda_a, *cuda_b, *cuda_c, *cuda_e;
	clock_t* time;

	hipMalloc((void**)&cuda_a, sizeof(float) * n * n);
	hipMalloc((void**)&cuda_b, sizeof(float) * n * n);
	hipMalloc((void**)&cuda_c, sizeof(float) * n * n);
	hipMalloc((void**)&time, sizeof(clock_t) * n * 2);

	hipMemcpy(cuda_a, a, sizeof(float) * n * n, hipMemcpyHostToDevice);
	hipMemcpy(cuda_b, b, sizeof(float) * n * n, hipMemcpyHostToDevice);

	//matMultCUDA<<<blocks_num, THREAD_NUM, 0>>>(cuda_a, cuda_b, cuda_c, n, time);
	matMultCUDA<<<n, THREAD_NUM, sizeof(float) * n >>> (cuda_a, cuda_b, cuda_c, n, time);

	clock_t time_use[blocks_num * 2];

	hipMemcpy(c, cuda_c, sizeof(float) * n * n, hipMemcpyDeviceToHost);
	hipMemcpy(&time_use, time, sizeof(clock_t) * 2 * blocks_num, hipMemcpyDeviceToHost);

	hipFree(cuda_a);
	hipFree(cuda_b);
	hipFree(cuda_c);
	hipFree(time);
	printf("sizeof(c)= %d\n",sizeof(c));

	for(int i = 0; i < 10; i++)
		printf("%f ", c[i]);
	
	clock_t min_start, max_end;
	min_start = time_use[0];
	max_end = time_use[blocks_num];
	for(int i = 1; i < blocks_num; i++){
		if(min_start > time_use[i])
			min_start = time_use[i];
		if(max_end < time_use[i + blocks_num])
			max_end = time_use[i + blocks_num];
	}
	clock_t  final_time = max_end - min_start;
	for(int i = 0; i < n; i++){
		for(int j = 0; j < n; j++){
			double t = 0;
			for(int k = 0; k < n; k++){
				t += a[i * n + k] * b[k * n + j];
			}
			d[i * n + j] = t;
		}
	}

	float max_err = 0;
	float average_err = 0;
	for(int i = 0; i < n; i++){
		for(int j = 0; j < n; j++){
			if(d[i * n + j] != 0){
				float err = fabs((c[i * n + j] - d[i * n + j]) / d[i * n + j]);
				if(max_err < err)
					max_err = err;
				average_err += err;
			}
		}
	}
	printf("Max error: %g Average error: %g\n", max_err, average_err / (n*n));

	printf("gputime: %d\n", final_time);
	return 0;
}
