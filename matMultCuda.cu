#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//CUDA RunTime API
#include <hip/hip_runtime.h>

#define THREAD_NUM 256

#define MATRIX_SIZE 1000

//int blocks_num = MATRIX_SIZE * MATRIX_SIZE / 1024;;
//const int blocks_num = MATRIX_SIZE*(MATRIX_SIZE + THREAD_NUM - 1) / THREAD_NUM;

//打印设备信息
void printDeviceProp(const hipDeviceProp_t &prop)
{
	printf("Device Name : %s.\n", prop.name);
	printf("totalFlbalMem : %d.\n", prop.totalGlobalMem);
	printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
	printf("regsPerBlock : %d.\n", prop.regsPerBlock);
	printf("warpSize : %d.\n", prop.warpSize);
	printf("memPitch : %d.\n", prop.memPitch);
}

bool InitCUDA()
{
	int count;

	//取得CUDA装置的数目
	hipGetDeviceCount(&count);

	if(count == 0){
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;

	for(i = 0; i < count; i ++){

		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		if(hipGetDeviceProperties(&prop, i) == hipSuccess){
			if(prop.major >= 1){
				break;
			}
		}
	}

	if(i == count){
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);
	return true;
}

void matgen(float* a, int n)
{
	int i, j;
	for(i = 0; i < n; i++){
		for(j = 0; j < n; j++)
		{
			a[i * n + j] = (float)rand() / RAND_MAX + (float)rand() / (RAND_MAX * RAND_MAX);
		}
	}
}

__global__ static void matMultCUDA(const float* a, const float* b, float* c, int n, clock_t* time, const int* blocks_num)
{
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	
	const int idx = bid * THREAD_NUM + tid;
	const int row = idx / n;
	const int column = idx % n;
	
	int i;
	
	 clock_t start;
	 
	 if(tid == 0) 
		 time[bid] = clock();
		 
	if(row < n && column < n){
		float t = 0;
		float y = 0;
		for(i = 0; i < n; i++){
			float r;
			y -= a[row * n + i] * b[i * n + column];
			r = t - y;
			y = (r-t)+y;
			t = r;
		}
		c[row * n + column] = t;
	}
	
	if(tid == 0){
		time[bid + *blocks_num] = clock();
	}
}

int func(int blocks_num)
{
	printf("%d\n", blocks_num);
	if(!InitCUDA()){
		return 0;
	}

	float *a, *b, *c, *d;
	int *e;
	
	int n = MATRIX_SIZE;

	a = (float*)malloc(sizeof(float)* n * n);
	b = (float*)malloc(sizeof(float)* n * n);
	c = (float*)malloc(sizeof(float)* n * n);
	d = (float*)malloc(sizeof(float)* n * n);
	e = (int*)malloc(sizeof(int));
	*e = blocks_num;
	srand(0);

	matgen(a, n);
	matgen(b, n);

	float *cuda_a, *cuda_b, *cuda_c;
	int *cuda_e;
	clock_t* time;

	hipMalloc((void**)&cuda_a, sizeof(float) * n * n);
	hipMalloc((void**)&cuda_b, sizeof(float) * n * n);
	hipMalloc((void**)&cuda_c, sizeof(float) * n * n);
	hipMalloc((void**)&cuda_e, sizeof(int));
	hipMalloc((void**)&time, sizeof(clock_t) * blocks_num * 2);

	hipMemcpy(cuda_a, a, sizeof(float) * n * n, hipMemcpyHostToDevice);
	hipMemcpy(cuda_b, b, sizeof(float) * n * n, hipMemcpyHostToDevice);
	hipMemcpy(cuda_e, e, sizeof(int), hipMemcpyHostToDevice);
	matMultCUDA<<<blocks_num, THREAD_NUM, 0>>>(cuda_a, cuda_b, cuda_c, n, time, cuda_e);

	clock_t time_use[blocks_num * 2];

	hipMemcpy(c, cuda_c, sizeof(float) * n * n, hipMemcpyDeviceToHost);
	hipMemcpy(&time_use, time, sizeof(clock_t) * blocks_num * 2, hipMemcpyHostToDevice);

	hipFree(cuda_a);
	hipFree(cuda_b);
	hipFree(cuda_c);
	hipFree(time);

	clock_t min_start, max_end;
	min_start = time_use[0];
	max_end = time_use[blocks_num];
	for(int i = 1; i < blocks_num; i++){
		if(min_start > time_use[i]){
			min_start = time_use[i];
		}
		if(max_end < time_use[i + blocks_num]){
			max_end = time_use[i + blocks_num];
		}
	}

	clock_t final_time = max_end - min_start;

	//CPU矩阵乘法，存入d中
	for(int i = 0; i < n; i++){
		for(int j = 0; j < n; j++){
			double t = 0;
			for(int k = 0; k < n; k++){
				t += a[i * n + k] * b[k * n + j];
			}
			d[i * n + j] = t;
		}
	}

	float max_err = 0;
	float average_err = 0;

	for(int i = 0; i < n; i++){
		for(int j = 0; j < n; j++){
			if(d[i * n + j] != 0){
				float err = fabs((c[i * n + j] - d[i * n + j]) / d[i * n + j]);

				if(max_err < err) 
					max_err = err;
					average_err += err;
			}
		}
	}

	printf("Max error: %g Average error: %g\n", max_err, average_err / (n * n));

	printf("gputime: %d\n", final_time);

	return 0;
}


int main()
{
	int blocks_num;
	//blocks_num = MATRIX_SIZE*(MATRIX_SIZE + THREAD_NUM - 1) / THREAD_NUM;
	//func(blocks_num);
	blocks_num = MATRIX_SIZE * MATRIX_SIZE / 1024;;
	func(blocks_num);
	//blocks_num = MATRIX_SIZE*(MATRIX_SIZE + THREAD_NUM - 1) / THREAD_NUM;
	//func(blocks_num);
}
